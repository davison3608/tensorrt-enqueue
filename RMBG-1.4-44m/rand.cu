using namespace nvinfer1;

BaseRand::BaseRand()
{
    hiprandCreateGenerator(&this->generator, HIPRAND_RNG_PSEUDO_DEFAULT);
}
BaseRand::~BaseRand()
{
    hiprandDestroyGenerator(this->generator);
}

//
UniformRand::UniformRand(Dims MatrixShape)
{
    this->MatrixShape=MatrixShape;
    this->nbdims=MatrixShape.nbDims;
    assert(this->nbdims != 0);
    
    this->singledim_vec.resize(this->nbdims);
        for (int i=0; i<this->singledim_vec.size(); i++)
    {
        assert(MatrixShape.d[i] != 0);
        this->singledim_vec[i]=MatrixShape.d[i];
    }
}
UniformRand::~UniformRand()
{}

void UniformRand::GenerateMatrixRand(std::vector<float>& outputmatrix)
{
    hipStream_t rand_str;
    hipStreamCreateWithFlags(&rand_str, hipStreamNonBlocking);
    //绑定流
    hiprandSetStream(this->generator, rand_str);
    //随机种子设定
    hiprandSetPseudoRandomGeneratorSeed(this->generator, 1234ULL);
    
    //infos
    float *matrix_ptr;
    std::size_t matrix_size=1;
    std::int32_t matrix_elements=1;
        for (int i=0; i<this->nbdims; i++)
    {
        matrix_elements=matrix_elements * this->singledim_vec[i];
    }
    matrix_size=matrix_elements * sizeof(float);
    //
    CUDA_CHECK(hipMallocAsync(&matrix_ptr, matrix_size, rand_str));
    CUDA_CHECK(hipMemset(matrix_ptr, 0, matrix_size));
    //
    hipStreamSynchronize(rand_str);
    //
    //随机生成0-1分布数
    auto status=hiprandGenerateUniform(this->generator, matrix_ptr, matrix_elements);
    assert(status == HIPRAND_STATUS_SUCCESS);
    //
    float *host_matrix_ptr;
    host_matrix_ptr=new float[matrix_elements];

    CUDA_CHECK(hipMemcpyAsync(host_matrix_ptr, matrix_ptr, matrix_size, hipMemcpyDeviceToHost, rand_str));
    hipStreamSynchronize(rand_str);

    if (outputmatrix.size() < matrix_elements)
        outputmatrix.resize(matrix_elements);
    //复制到容器
    std::copy(host_matrix_ptr, host_matrix_ptr + matrix_elements, outputmatrix.begin());
    assert(outputmatrix.size() == matrix_elements);
    //
    delete[] host_matrix_ptr;
    hipFree(matrix_ptr);
    return ;
}
