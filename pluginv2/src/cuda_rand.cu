#include "../include/cuda_rand.h"
using namespace nvinfer1;

BaseRand::BaseRand()
{
    hiprandCreateGenerator(&this->generator, HIPRAND_RNG_PSEUDO_DEFAULT);
}
BaseRand::~BaseRand()
{
    hiprandDestroyGenerator(this->generator);
}

//
UniformRand::UniformRand(Dims MatrixShape)
{
    this->MatrixShape=MatrixShape;
    this->nbdims=MatrixShape.nbDims;
    assert(this->nbdims != 0);
    
    this->singledim_vec.resize(this->nbdims);
        for (int i=0; i<this->singledim_vec.size(); i++)
    {
        assert(MatrixShape.d[i] != 0);
        this->singledim_vec[i]=MatrixShape.d[i];
    }
}
UniformRand::~UniformRand()
{}

void UniformRand::GenerateMatrixRand(std::vector<float>& outputmatrix)
{
    hipStream_t rand_str;
    hipStreamCreateWithFlags(&rand_str, hipStreamNonBlocking);
    //绑定流
    hiprandSetStream(this->generator, rand_str);
    //随机种子设定
    hiprandSetPseudoRandomGeneratorSeed(this->generator, 1234ULL);
    
    //infos
    float *matrix_ptr;
    std::size_t matrix_size=1;
    std::int32_t matrix_elements=1;
        for (int i=0; i<this->nbdims; i++)
    {
        matrix_elements=matrix_elements * this->singledim_vec[i];
    }
    matrix_size=matrix_elements * sizeof(float);
    //
    CUDA_CHECK(hipMallocAsync(&matrix_ptr, matrix_size, rand_str));
    CUDA_CHECK(hipMemset(matrix_ptr, 0, matrix_size));
    //
    hipStreamSynchronize(rand_str);
    //
    //随机生成0-1分布数
    auto status=hiprandGenerateUniform(this->generator, matrix_ptr, matrix_elements);
    assert(status == HIPRAND_STATUS_SUCCESS);
    //
    float *host_matrix_ptr;
    host_matrix_ptr=new float[matrix_elements];

    CUDA_CHECK(hipMemcpyAsync(host_matrix_ptr, matrix_ptr, matrix_size, hipMemcpyDeviceToHost, rand_str));
    hipStreamSynchronize(rand_str);

    if (outputmatrix.size() < matrix_elements)
        outputmatrix.resize(matrix_elements);
    //复制到容器
    std::copy(host_matrix_ptr, host_matrix_ptr + matrix_elements, outputmatrix.begin());
    assert(outputmatrix.size() == matrix_elements);
    //
    delete[] host_matrix_ptr;
    hipFree(matrix_ptr);
    return ;
}

//
NormalRand::NormalRand(Dims MatrixShape)
{
    this->MatrixShape=MatrixShape;
    this->nbdims=MatrixShape.nbDims;
    assert(this->nbdims != 0);
    
    this->singledim_vec.resize(this->nbdims);
        for (int i=0; i<this->singledim_vec.size(); i++)
    {
        this->singledim_vec[i]=MatrixShape.d[i];
    }
}
NormalRand::~NormalRand()
{}

void NormalRand::GenerateMatrixRand(std::vector<float>& outputmatrix)
{
    hipStream_t rand_str;
    hipStreamCreateWithFlags(&rand_str, hipStreamNonBlocking);
    //绑定流
    hiprandSetStream(this->generator, rand_str);
    //随机种子设定
    hiprandSetPseudoRandomGeneratorSeed(this->generator, 1234ULL);
    
    //infos
    float *matrix_ptr;
    std::size_t matrix_size=1;
    std::int32_t matrix_elements=1;
        for (int i=0; i<this->nbdims; i++)
    {
        matrix_elements=matrix_elements * this->singledim_vec[i];
    }
    matrix_size=matrix_elements * sizeof(float);
    //
    CUDA_CHECK(hipMallocAsync(&matrix_ptr, matrix_size, rand_str));
    CUDA_CHECK(hipMemset(matrix_ptr, 0, matrix_size));
    //
    hipStreamSynchronize(rand_str);
    //
    //随机正态度分布生成
    int n=matrix_elements;
    float mean=0.0f;
    float stddev=1.0f;
    auto status=hiprandGenerateNormal(this->generator, matrix_ptr, n, mean, stddev);
    assert(status == HIPRAND_STATUS_SUCCESS);
    //
    float *host_matrix_ptr;
    host_matrix_ptr=new float[matrix_elements];

    CUDA_CHECK(hipMemcpyAsync(host_matrix_ptr, matrix_ptr, matrix_size, hipMemcpyDeviceToHost, rand_str));
    hipStreamSynchronize(rand_str);

    if (outputmatrix.size() < matrix_elements)
        outputmatrix.resize(matrix_elements);
    //复制到容器
    std::copy(host_matrix_ptr, host_matrix_ptr + matrix_elements, outputmatrix.begin());
    assert(outputmatrix.size() == matrix_elements);
    //
    delete[] host_matrix_ptr;
    hipFree(matrix_ptr);
    return ;
}

//
PoissonRand::PoissonRand(Dims MatrixShape)
{
    this->MatrixShape=MatrixShape;
    this->nbdims=MatrixShape.nbDims;
    assert(this->nbdims != 0);
    
    this->singledim_vec.resize(this->nbdims);
        for (int i=0; i<this->singledim_vec.size(); i++)
    {
        this->singledim_vec[i]=MatrixShape.d[i];
    }
}
PoissonRand::~PoissonRand()
{}

void PoissonRand::GenerateMatrixRand(std::vector<float>& outputmatrix) 
{}    
void PoissonRand::GenerateMatrixRand(std::vector<uint>& outputmatrix) 
{
    hipStream_t rand_str;
    hipStreamCreateWithFlags(&rand_str, hipStreamNonBlocking);
    //绑定流
    hiprandSetStream(this->generator, rand_str);
    //随机种子设定
    hiprandSetPseudoRandomGeneratorSeed(this->generator, 1234ULL);
    
    //infos
    uint *matrix_ptr;
    std::size_t matrix_size=1;
    std::int32_t matrix_elements=1;
        for (int i=0; i<this->nbdims; i++)
    {
        matrix_elements=matrix_elements * this->singledim_vec[i];
    }
    matrix_size=matrix_elements * sizeof(uint);
    //
    CUDA_CHECK(hipMallocAsync(&matrix_ptr, matrix_size, rand_str));
    CUDA_CHECK(hipMemset(matrix_ptr, 0, matrix_size));
    //
    hipStreamSynchronize(rand_str);
    
    //柏松期望值
    double lambda=5.0;
    auto status=hiprandGeneratePoisson(this->generator, matrix_ptr, matrix_elements, lambda);
    assert(status);
    //
    uint *host_matrix_ptr;
    host_matrix_ptr=new uint[matrix_elements];

    CUDA_CHECK(hipMemcpyAsync(host_matrix_ptr, matrix_ptr, matrix_size, hipMemcpyDeviceToHost, rand_str));
    hipStreamSynchronize(rand_str);

    if (outputmatrix.size() < matrix_elements)
        outputmatrix.resize(matrix_elements);
    //复制到容器
    std::copy(host_matrix_ptr, host_matrix_ptr + matrix_elements, outputmatrix.begin());
    assert(outputmatrix.size() == matrix_elements);
    //
    delete[] host_matrix_ptr;
    hipFree(matrix_ptr);
    return ;
}

//
IntRand::IntRand(Dims MatrixShape)
{
    this->MatrixShape=MatrixShape;
    this->nbdims=MatrixShape.nbDims;
    assert(this->nbdims != 0);
    
    this->singledim_vec.resize(this->nbdims);
        for (int i=0; i<this->singledim_vec.size(); i++)
    {
        this->singledim_vec[i]=MatrixShape.d[i];
    }
}
IntRand::~IntRand()
{}

void IntRand::GenerateMatrixRand(std::vector<float>& outputmatrix) 
{} 
void IntRand::GenerateMatrixRand(std::vector<uint>& outputmatrix)
{
    hipStream_t rand_str;
    hipStreamCreateWithFlags(&rand_str, hipStreamNonBlocking);
    //绑定流
    hiprandSetStream(this->generator, rand_str);
    //随机种子设定
    hiprandSetPseudoRandomGeneratorSeed(this->generator, 1234ULL);
    
    //infos
    uint *matrix_ptr;
    std::size_t matrix_size=1;
    std::int32_t matrix_elements=1;
        for (int i=0; i<this->nbdims; i++)
    {
        matrix_elements=matrix_elements * this->singledim_vec[i];
    }
    matrix_size=matrix_elements * sizeof(uint);
    //
    CUDA_CHECK(hipMallocAsync(&matrix_ptr, matrix_size, rand_str));
    CUDA_CHECK(hipMemset(matrix_ptr, 0, matrix_size));
    //
    hipStreamSynchronize(rand_str);
    //
    auto status=hiprandGenerate(this->generator, matrix_ptr, matrix_elements);
        //
    uint *host_matrix_ptr;
    host_matrix_ptr=new uint[matrix_elements];

    CUDA_CHECK(hipMemcpyAsync(host_matrix_ptr, matrix_ptr, matrix_size, hipMemcpyDeviceToHost, rand_str));
    hipStreamSynchronize(rand_str);

    if (outputmatrix.size() < matrix_elements)
        outputmatrix.resize(matrix_elements);
    //复制到容器
    std::copy(host_matrix_ptr, host_matrix_ptr + matrix_elements, outputmatrix.begin());
    assert(outputmatrix.size() == matrix_elements);
    //
    delete[] host_matrix_ptr;
    hipFree(matrix_ptr);
    return ;
}
////
//LogNormalRand::LogNormalRand(Dims MatrixShape)
//{
//    this->MatrixShape=MatrixShape;
//    this->nbdims=MatrixShape.nbDims;
//    assert(this->nbdims != 0);
//    
//    this->singledim_vec.resize(this->nbdims);
//        for (int i=0; i<this->singledim_vec.size(); i++)
//    {
//        this->singledim_vec[i]=MatrixShape.d[i];
//    }
//}
//LogNormalRand::~LogNormalRand()
//{}







































